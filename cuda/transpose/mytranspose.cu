/*
a4230755 矩陣轉置

*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
void printMat(const float *mat, const int row, const int col);
void randomMat(float *mat, const int lenght);
__global__ void transpose_gpu(float *mat, float *result);

int main(){
    int row = 10;
    int col = 5;
    float *mat = (float*) malloc(sizeof(float)*row*col);
    float *result = (float*) malloc(sizeof(float)*row*col);
    float *device_mat, *device_result;

    //generate random matrix and print its
    randomMat(mat,row*col);
    printf("pre-transpose : \n");
    printMat(mat,row,col);
    
    hipMalloc((void**)&device_mat, sizeof(float)*row*col);
    hipMalloc((void**)&device_result, sizeof(float)*row*col);
    hipMemcpy(device_mat, mat, sizeof(float)*row*col, hipMemcpyHostToDevice);

    //allocation gpu resource
    dim3 grid(row,1,1),block(col,1,1);
    //transpose matrix on gpu
    transpose_gpu<<<grid,block>>>(device_mat,device_result);
    hipMemcpy(result, device_result, sizeof(float)*row*col, hipMemcpyDeviceToHost);
    printf("transposed : \n");
    printMat(result,col,row);

}

__global__ void transpose_gpu(float *mat, float *result){
    int row = blockIdx.x;
    int col = threadIdx.x;
    result[col*gridDim.x + row] = mat[blockDim.x*row+col];
}

void printMat(const float *mat, const int row, const int col){
    for(int i=0; i< row*col; i++){
        printf("%f ", mat[i]);
        if((i+1) % col == 0){
            printf("\n");
        }
    }
};

void randomMat(float *mat, const int lenght){
    srand(time(NULL));
    for(int i=0; i< lenght; i++){
        mat[i] = (float) rand()/ (float)(RAND_MAX) * 1.0;
    }
};