
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
#include "time.h"

__global__ void matMultiply_Gpu(int *a, int *b, int *c, int width){
    int row = threadIdx.x;
    int col = threadIdx.y;
    int partial_ans = 0;
    if(row < width && col < width){
        for(int n=0; n< width; n++){
            //sigma 0 to n > c(i,j) = a[i,n]*b[n,j];
            partial_ans = a[row*width+n] * b[n*width+col];
            c[row*width+col] += partial_ans;
        }
    }
}

//隨機產生矩陣
void intMatrixGen(int *mat, int length, char type){
    int i,j;
    for(i=0;i<length;i++){
        for(j=0;j<length;j++){
            switch(type){
                case 'r':
                    mat[i*length+j] = (rand() % 20)+1;    
                    break;
                case 0:
                    mat[i*length+j] = 0;
                    break;
                default:
                    mat[i*length+j] = 0;
            }
        }
    }
};



//印出矩陣
void printIntMatrix(int *mat,int length){
    int i,j;

    for(i=0;i<length;i++){
        for(j=0;j<length;j++){

            printf("%d ",mat[i*length+j]);
        }
        printf("\n");
    }

    printf("\n");
};

void matMultiply_Cpu(int *a, int *b, int *c, int length){
    for(int i=0; i< length; ++i){
        for(int j=0; j< length; ++j){
            for(int k=0; k< length; ++k){
                c[i*length+j] += a[i*length+k]*b[k*length+j];
            }
        }
    }
}

int main(){
    int length =3;
    //設定seed
    unsigned seed;
    seed = (unsigned)time(NULL); // 取得時間序列
    srand(seed); // 以時間序列當亂數種子

    //初始化矩陣
    int *a = (int*) malloc(sizeof(int) * length * length);
    int *b = (int*) malloc(sizeof(int) * length * length);
    int *result = (int*) malloc(sizeof(int) * length * length);
    int *result_cpu = (int*) malloc(sizeof(int) * length * length);

    //產生矩陣
    intMatrixGen(a,length,'r');
    intMatrixGen(b,length,'r');
    intMatrixGen(result_cpu,length, 0);
    printIntMatrix(a,length);
    printIntMatrix(b,length);
    //分配資源
    int *gpu_a,*gpu_b,*gpu_result;
    hipMalloc((void**)&gpu_a, length * length * sizeof(int));      
    hipMalloc((void**)&gpu_b, length * length * sizeof(int));
    hipMalloc((void**)&gpu_result, length * length * sizeof(int));
    //複製資料到gpu
    hipMemcpy(gpu_a, a, length *length * sizeof(int), hipMemcpyHostToDevice);    
    hipMemcpy(gpu_b, b, length *length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_result, result, 0, hipMemcpyHostToDevice);
    //開始計算
    dim3 grid(1,1,1), block(length,length,1);
    matMultiply_Gpu<<<grid,block>>>(gpu_a,gpu_b,gpu_result,length);
    //複製回cpu
    hipMemcpy(result, gpu_result, length * length * sizeof(int), hipMemcpyDeviceToHost);
    printf("gpu result:\n");
    printIntMatrix(result,length);
    
    matMultiply_Cpu(a,b,result_cpu,length);
    printf("cpu result:\n");
    printIntMatrix(result_cpu,length);

    return 0;
}